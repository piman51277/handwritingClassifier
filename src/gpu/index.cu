#include <iostream>
#include <fstream>
#include <cinttypes>
#include <memory>
#include <random>
#include <vector>
#include "matrix.h"
#include "mnist.h"
#include "net.h"

#include <chrono>
typedef std::chrono::high_resolution_clock Clock;

void visualizeMNIST(TrainingData &data, uint32_t index)
{
  Matrix &input = data.input;
  Matrix &expected = data.expected;

  // copy data to host
  double *inputData;
  double *expectedData;
  hipHostMalloc(&inputData, input.dim1 * input.dim2 * sizeof(double));
  hipHostMalloc(&expectedData, expected.dim1 * expected.dim2 * sizeof(double));
  hipMemcpy(inputData, input.mat.get(), input.dim1 * input.dim2 * sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(expectedData, expected.mat.get(), expected.dim1 * expected.dim2 * sizeof(double), hipMemcpyDeviceToHost);

  // images are 28x28, and we are looking at the index-th column
  for (uint32_t i = 0; i < 28; i++)
  {
    for (uint32_t j = 0; j < 28; j++)
    {
      if (inputData[index + (i * 28 + j) * input.dim2] > 0.1)
      {
        std::cout << "X";
      }
      else
      {
        std::cout << "-";
      }
    }
    std::cout << std::endl;
  }

  // print expected label
  for (uint32_t i = 0; i < 10; i++)
  {
    if (expectedData[index + i * expected.dim2] > 0.1)
    {
      std::cout << i << std::endl;
    }
  }
}

std::vector<uint32_t> getSettings(const char *filename)
{
  std::ifstream file(filename, std::ios::binary);
  int layers;
  file >> layers;
  std::cout << "Layers: " << layers << std::endl;
  std::vector<uint32_t> settings(layers);
  for (int i = 0; i < layers; i++)
  {
    std::cout << "Layer " << i << ": ";
    file >> settings[i];
    std::cout << settings[i] << std::endl;
  }
  return settings;
}

void train(const char *filename)
{
  hipDeviceReset();

  std::cout << "Loading MNIST data..." << std::endl;
  TrainingDataSet MNIST = get_MNIST("./data/train-images-idx3-ubyte", "./data/train-labels-idx1-ubyte", 4000);
  TrainingDataSet MNIST_test = get_MNIST("./data/t10k-images-idx3-ubyte", "./data/t10k-labels-idx1-ubyte", 2000);
  std::cout << "MNIST data loaded." << std::endl;

  // create network
  std::vector<uint32_t> layers = getSettings("config.txt");
  Net net(layers);
  net.initializeWeights();
  std::cout << "Network created." << std::endl;

  // test network
  std::cout << "Testing network..." << std::endl;
  double error = net.error(MNIST_test);
  std::cout << "Initial error (MSE): " << error << std::endl;
  error = net.error_percent(MNIST_test);
  std::cout << "Initial accuracy (percent): " << error << std::endl;

  // train network
  std::cout << "Training network..." << std::endl;
  TrainConfig config{1000, 4000, 0.2, 0.2, 0.95};

  auto t1 = Clock::now();
  net.train(MNIST, config);
  auto t2 = Clock::now();

  std::cout << "Training complete." << std::endl;
  std::cout << "Training time: " << std::chrono::duration_cast<std::chrono::milliseconds>(t2 - t1).count() << " ms" << std::endl;

  // test network
  std::cout << "Testing network..." << std::endl;
  error = net.error(MNIST_test);
  std::cout << "Final error (MSE): " << error << std::endl;
  error = net.error_percent(MNIST_test);
  std::cout << "Final error (percent): " << error << std::endl;

  // save the net to disk
  net.save(filename);
}

int main()
{
  train("network.bin");

  TrainingDataSet MNIST_test = get_MNIST("./data/t10k-images-idx3-ubyte", "./data/t10k-labels-idx1-ubyte", 1000);

  // load the net from disk
  std::cout << "Loading saved network..." << std::endl;
  Net net2 = Net::load("network.bin");

  // test network
  std::cout << "Testing network..." << std::endl;
  double error = net2.error(MNIST_test);
  std::cout << "Final error (MSE): " << error << std::endl;
  error = net2.error_percent(MNIST_test);
  std::cout << "Final error (percent): " << error << std::endl;
}